#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << " line " << __LINE__ \
                      << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

const size_t MEMORY_SIZE = 20ULL * 1024 * 1024 * 1024;  // 20GB

void testMemoryCopy(int deviceId, void* h_memory) {
    hipSetDevice(deviceId);
    
    void *d_memory;
    
    // 分配设备内存
    CHECK_CUDA(hipMalloc(&d_memory, MEMORY_SIZE));
    
    clock_t start, end;
    double cpu_time_used;
    
    // 主机到设备的拷贝
    start = clock();
    CHECK_CUDA(hipMemcpy(d_memory, h_memory, MEMORY_SIZE, hipMemcpyHostToDevice));
    end = clock();
    
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    double speed_h2d = MEMORY_SIZE / cpu_time_used / (1024.0 * 1024 * 1024);
    
    // 设备到主机的拷贝
    start = clock();
    CHECK_CUDA(hipMemcpy(h_memory, d_memory, MEMORY_SIZE, hipMemcpyDeviceToHost));
    end = clock();
    
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    double speed_d2h = MEMORY_SIZE / cpu_time_used / (1024.0 * 1024 * 1024);
    
    std::cout << "Device " << deviceId << ":" << std::endl;
    std::cout << "  Host to Device: " << speed_h2d << " GB/s" << std::endl;
    std::cout << "  Device to Host: " << speed_d2h << " GB/s" << std::endl;
    
    // 释放设备内存
    CHECK_CUDA(hipFree(d_memory));
}

int main() {
    int deviceCount;
    CHECK_CUDA(hipGetDeviceCount(&deviceCount));
    
    std::cout << "Found " << deviceCount << " CUDA devices" << std::endl;

    // 只分配一次主机内存
    void *h_memory;
    CHECK_CUDA(hipHostMalloc(&h_memory, MEMORY_SIZE, hipHostMallocDefault));
    
    for (int i = 0; i < deviceCount && i < 4; ++i) {
        testMemoryCopy(i, h_memory);
    }
    
    // 释放主机内存
    CHECK_CUDA(hipHostFree(h_memory));
    
    return 0;
}
